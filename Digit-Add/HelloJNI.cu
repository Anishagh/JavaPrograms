#include "hip/hip_runtime.h"
#include <jni.h>        
#include <stdio.h>      
#include <math.h>
#include <stdlib.h>
#include "HelloJNI.h"
 

__global__ void add(jint *x, jint*y, jint *r, jint sz) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x; //Getting thread number
	if(i<sz)
	{
   	jint z=((int)(*x/pow(10,i))%10)+((int)(*y/pow(10,i))%10); //Adding digits
	r[i]=z;	
	}  
}

void show(jint* r, jint sz)
{
	for(int i=sz-1;i>=0;i--)
	{
		printf("%d ",r[i]); //Showing number
	}
}

JNIEXPORT void JNICALL Java_HelloJNI_sayHello(JNIEnv *env, jobject thisObj, jint a, jint b) {
	jint sz=(int)log10(max(a,b))+1;
	jint* r;
	hipMallocManaged(&r,sz*sizeof(jint));
	jint *a1;
	jint *b1;
	hipMallocManaged(&a1,sizeof(jint));
	hipMallocManaged(&b1,sizeof(jint));
	hipMemcpy(a1,&a,sizeof(jint),hipMemcpyHostToDevice);
	hipMemcpy(b1,&b,sizeof(jint),hipMemcpyHostToDevice);
	add<<<1,sz>>> (a1,b1,r,sz); //Parallel addition of digits
	hipDeviceSynchronize();
	show(r,sz);
}
